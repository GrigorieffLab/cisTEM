#include "hip/hip_runtime.h"
#include "gpu_core_headers.h"

#define DO_HISTOGRAM true

__global__ void UpdateMipPixelWiseKernel(__half* correlation_output, __half2* my_peaks, const int numel, __half psi, __half theta, __half phi, __half2* my_stats, __half2* my_new_peaks, const int view_counter);

TemplateSnrRatioCore::TemplateSnrRatioCore( ){

};

TemplateSnrRatioCore::TemplateSnrRatioCore(int number_of_jobs) {

    Init(number_of_jobs);
};

TemplateSnrRatioCore::~TemplateSnrRatioCore( ){

        // FIXME
        //	if (is_allocated_cummulative_histogram)
        //	{
        //		cudaErr(hipFree((void *)cummulative_histogram));
        //		cudaErr(hipHostFree((void *)h_cummulative_histogram));
        //	}

};

void TemplateSnrRatioCore::Init(int number_of_jobs) {
    this->nThreads                        = 1;
    this->number_of_jobs_per_image_in_gui = 1;
    this->nGPUs                           = 1;
};

void TemplateSnrRatioCore::Init(MyApp*           parent_pointer,
                                Image&           input_reconstruction_particle,
                                Image&           input_reconstruction_correct,
                                Image&           input_reconstruction_wrong,
                                Image&           input_image,
                                Image&           current_projection_image,
                                Image&           current_projection_correct_template,
                                Image&           current_projection_other,
                                float            psi_max,
                                float            psi_start,
                                float            psi_step_sampled_view,
                                float            psi_step_tm,
                                AnglesAndShifts& angles_sampled_view,
                                AnglesAndShifts& angles_tm,
                                EulerSearch&     global_euler_search_sampled_view,
                                EulerSearch&     global_euler_search_tm,
                                int              first_search_position_sampled_view,
                                int              last_search_position_sampled_view,
                                int              first_search_position_tm,
                                int              last_search_position_tm,
                                ProgressBar*     my_progress,
                                int              number_of_rotations_sampled_view,
                                long             total_correlation_positions_sampled_view,
                                long             total_correlation_positions_sampled_view_per_thread,
                                float            avg_for_normalization,
                                float            std_for_normalization)

{
    this->first_search_position_sampled_view = first_search_position_sampled_view;
    this->first_search_position_tm           = first_search_position_tm;
    this->last_search_position_sampled_view  = last_search_position_sampled_view;
    this->last_search_position_tm            = last_search_position_tm;
    this->angles_sampled_view                = angles_sampled_view;
    this->angles_tm                          = angles_tm;
    this->global_euler_search_sampled_view   = global_euler_search_sampled_view;
    this->global_euler_search_tm             = global_euler_search_tm;

    this->psi_start             = psi_start;
    this->psi_step_sampled_view = psi_step_sampled_view;
    this->psi_step_tm           = psi_step_tm;
    this->psi_max               = psi_max;

    this->avg_for_normalization = avg_for_normalization;
    this->std_for_normalization = std_for_normalization;

    this->number_of_rotations_sampled_view                    = number_of_rotations_sampled_view;
    this->total_correlation_positions_sampled_view            = total_correlation_positions_sampled_view;
    this->total_correlation_positions_sampled_view_per_thread = total_correlation_positions_sampled_view_per_thread;

    // 3D volumes for particle, correct template and incorrect template
    this->input_reconstruction_correct.CopyFrom(&input_reconstruction_correct);
    this->input_reconstruction_wrong.CopyFrom(&input_reconstruction_wrong);
    this->input_reconstruction_particle.CopyFrom(&input_reconstruction_particle);

    // this->input_image.CopyFrom(&input_image);
    this->current_projection_image.CopyFrom(&current_projection_image);
    this->current_projection_other.CopyFrom(&current_projection_other);
    this->current_projection_correct_template.CopyFrom(&current_projection_correct_template);

    // projections can be created on GPU only, no transfer from / to host
    // FIXME & TODO: figure out what should be copied to device and what should be created on device
    d_current_projection_image.Init(this->current_projection_image);
    d_current_projection_other.Init(this->current_projection_other);
    d_current_projection_correct_template.Init(this->current_projection_correct_template);
    d_padded_image.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, 1, true); // TODO may need to include padding
    d_padded_reference_correct.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, 1, true); // TODO may need to include padding
    d_padded_reference_wrong.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, 1, true); // TODO may need to include padding

    // d_input_image.Init(this->input_image);
    // d_input_image.CopyHostToDevice( );
    //d_padded_reference.Allocate(d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);
    d_max_intensity_projection_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);

    d_max_intensity_projection_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);

    // d_best_psi.Allocate(d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);
    // d_best_theta.Allocate(d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);
    // d_best_phi.Allocate(d_input_image.dims.x, d_input_image.dims.y, d_input_image.dims.z, true);

    d_sum1_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sum1_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sum2_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sum2_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sum3_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sum3_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);

    d_sumSq1_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sumSq1_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sumSq2_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sumSq2_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sumSq3_ac.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);
    d_sumSq3_cc.Allocate(this->input_reconstruction_particle.logical_x_dimension, this->input_reconstruction_particle.logical_y_dimension, int(this->total_correlation_positions_sampled_view), true);

    this->my_progress = my_progress;

    this->parent_pointer = parent_pointer;

    // For now we are only working on the inner loop, so no need to track best_defocus and best_pixel_size

    // At the outset these are all empty cpu images, so don't xfer, just allocate on gpuDev

    // Transfer the input image_memory_should_not_be_deallocated

    cudaErr(hipStreamSynchronize(hipStreamPerThread));
};

void TemplateSnrRatioCore::RunInnerLoop(Image& projection_filter, int threadIDX, long& current_correlation_position_sampled_view) {

    // Make sure we are starting with zeros
    d_max_intensity_projection_ac.SetToConstant(-FLT_MAX);
    d_max_intensity_projection_cc.SetToConstant(-FLT_MAX);

    d_padded_image.Zeros( );
    d_padded_image.ConvertToHalfPrecision(false);

    d_padded_reference_correct.Zeros( );
    d_padded_reference_correct.ConvertToHalfPrecision(false);

    d_padded_reference_wrong.Zeros( );
    d_padded_reference_wrong.ConvertToHalfPrecision(false);

    //d_best_psi.Zeros( );
    //d_best_phi.Zeros( );
    //d_best_theta.Zeros( );

    d_sum1_ac.Zeros( );
    d_sumSq1_ac.Zeros( );
    d_sum2_ac.Zeros( );
    d_sumSq2_ac.Zeros( );
    d_sum3_ac.Zeros( );
    d_sumSq3_ac.Zeros( );

    d_sum1_cc.Zeros( );
    d_sumSq1_cc.Zeros( );
    d_sum2_cc.Zeros( );
    d_sumSq2_cc.Zeros( );
    d_sum3_cc.Zeros( );
    d_sumSq3_cc.Zeros( );

    total_number_of_cccs_calculated = 0;

    // Either do not delete the single precision, or add in a copy here so that each loop over defocus vals
    // have a copy to work with. Otherwise this will not exist on the second loop

    cudaErr(hipMalloc((void**)&my_peaks_ac, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_peaks_cc, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_new_peaks_ac, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_new_peaks_cc, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_stats_ac, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMalloc((void**)&my_stats_cc, sizeof(__half2) * total_correlation_positions_sampled_view * d_current_projection_image.real_memory_allocated));

    cudaErr(hipMemset(my_peaks_ac, 0, total_correlation_positions_sampled_view * sizeof(__half2) * d_current_projection_image.real_memory_allocated));
    cudaErr(hipMemset(my_peaks_cc, 0, total_correlation_positions_sampled_view * sizeof(__half2) * d_current_projection_image.real_memory_allocated));
    // cudaErr(hipMemset(my_stats_ac, 0, total_correlation_positions_sampled_view * sizeof(__half2) * d_current_projection_image.real_memory_allocated));
    // cudaErr(hipMemset(my_stats_cc, 0, total_correlation_positions_sampled_view * sizeof(__half2) * d_current_projection_image.real_memory_allocated));

    hipEvent_t image_projection_is_free_Event, ref_correct_projection_is_free_Event, ref_wrong_projection_is_free_Event, current_view_is_done,
            gpu_work_is_done_Event, current_tm_is_done;
    cudaErr(hipEventCreateWithFlags(&image_projection_is_free_Event, hipEventDisableTiming));
    cudaErr(hipEventCreateWithFlags(&ref_correct_projection_is_free_Event, hipEventDisableTiming));
    cudaErr(hipEventCreateWithFlags(&ref_wrong_projection_is_free_Event, hipEventDisableTiming));

    cudaErr(hipEventCreateWithFlags(&current_tm_is_done, hipEventDisableTiming));
    cudaErr(hipEventCreateWithFlags(&current_view_is_done, hipEventDisableTiming));

    int   ccc_counter;
    int   current_search_position_sampled_view, current_search_position_tm;
    float average_on_edge;
    float average_of_reals;
    float temp_float;
    float current_psi_tm;
    float variance;

    int thisDevice;
    hipGetDevice(&thisDevice);
    wxPrintf("Thread %d is running on device %d\n", threadIDX, thisDevice);
    int   view_counter;
    float current_psi_sampled_view;

    for ( current_search_position_sampled_view = first_search_position_sampled_view; current_search_position_sampled_view <= last_search_position_sampled_view; current_search_position_sampled_view++ ) {
        if ( current_search_position_sampled_view % 10 == 0 ) {
            wxPrintf("Starting position %d/ %d\n", current_search_position_sampled_view, last_search_position_sampled_view);
        }
        for ( int j = 0; j < number_of_rotations_sampled_view; j++ ) { // check if the max is actually the psi_max CHECKME FIXME
            ccc_counter              = 0; // stores the tm ccs calculated
            current_psi_sampled_view = psi_start + j * psi_step_sampled_view;
            view_counter             = current_search_position_sampled_view * number_of_rotations_sampled_view + j;
            wxPrintf("worker %i starts on view %i\n\n", ReturnThreadNumberOfCurrentThread( ), view_counter);
            angles_sampled_view.Init(global_euler_search_sampled_view.list_of_search_parameters[current_search_position_sampled_view][0], global_euler_search_sampled_view.list_of_search_parameters[current_search_position_sampled_view][1], current_psi_sampled_view, 0.0, 0.0);
            //			current_projection.SetToConstant(0.0f); // This also sets the FFT padding to zero

            input_reconstruction_particle.ExtractSlice(current_projection_image, angles_sampled_view, 1.0f, false);
            current_projection_image.SwapRealSpaceQuadrants( );
            current_projection_image.MultiplyPixelWise(projection_filter);
            current_projection_image.BackwardFFT( );

            current_projection_image.AddConstant(-avg_for_normalization);
            current_projection_image.DivideByConstant(std_for_normalization);
            current_projection_image.ForwardFFT( );
            current_projection_image.SwapRealSpaceQuadrants( );
            current_projection_image.QuickAndDirtyWriteSlice(wxString::Format("check_gpu_run/c_img_%i.mrc", view_counter).ToStdString( ), 1);

            hipStreamWaitEvent(hipStreamPerThread, image_projection_is_free_Event, 0);
            d_current_projection_image.CopyHostToDevice( );
            d_current_projection_image.QuickAndDirtyWriteSlices(wxString::Format("check_gpu_run/d_img_%i.mrc", view_counter).ToStdString( ), 1, 1);

            d_current_projection_image.ConvertToHalfPrecision(false);

            // normalize using avg and std from all sampled views
            // d_current_projection_image.AddConstant(-avg_for_normalization);
            // d_current_projection_image.MultiplyByConstant(1 / std_for_normalization);
            // d_current_projection_image.ClipInto(&d_padded_image, 0, false, 0, 0, 0, 0);

            //  d_padded_image.ForwardFFT(false); // IMPORTANT CHEKCME scaling must set to false ow the output is only hald of the input image
            // FIXME TODO move fft into cpu?
            // d_padded_image.SwapRealSpaceQuadrants( );

            for ( current_search_position_tm = first_search_position_tm; current_search_position_tm <= last_search_position_tm; current_search_position_tm++ ) {
                for ( current_psi_tm = psi_start; current_psi_tm <= psi_max; current_psi_tm += psi_step_tm ) {
                    angles_tm.Init(global_euler_search_tm.list_of_search_parameters[current_search_position_tm][0], global_euler_search_tm.list_of_search_parameters[current_search_position_tm][1], current_psi_tm, 0.0, 0.0);
                    // generate projection from testing template for tm
                    input_reconstruction_wrong.ExtractSlice(current_projection_other, angles_tm, 1.0f, false);
                    current_projection_other.SwapRealSpaceQuadrants( );
                    current_projection_other.MultiplyPixelWise(projection_filter);
                    current_projection_other.BackwardFFT( );
                    //average_on_edge  = current_projection_other.ReturnAverageOfRealValuesOnEdges( );
                    average_of_reals = current_projection_other.ReturnAverageOfRealValues( );
                    variance         = current_projection_other.ReturnSumOfSquares( ) - powf(current_projection_other.ReturnAverageOfRealValues( ), 2);
                    hipStreamWaitEvent(hipStreamPerThread, ref_wrong_projection_is_free_Event, 0);
                    //// TO THE GPU ////
                    d_current_projection_other.CopyHostToDevice( );
                    d_current_projection_other.AddConstant(-average_of_reals);
                    d_current_projection_other.MultiplyByConstant(rsqrtf(variance));
                    //current_projection_2.AddGaussianNoise(10.0f);
                    // Zeroing the central pixel is probably not doing anything useful...
                    // d_current_projection_other.ZeroCentralPixel( );

                    d_current_projection_other.ClipInto(&d_padded_reference_wrong, 0, false, 0, 0, 0, 0);
                    hipEventRecord(ref_wrong_projection_is_free_Event, hipStreamPerThread);

                    d_padded_reference_wrong.ForwardFFT(false);

                    input_reconstruction_correct.ExtractSlice(current_projection_correct_template, angles_tm, 1.0f, false);
                    current_projection_correct_template.SwapRealSpaceQuadrants( );
                    current_projection_correct_template.MultiplyPixelWise(projection_filter);
                    current_projection_correct_template.BackwardFFT( );

                    average_of_reals = current_projection_correct_template.ReturnAverageOfRealValues( );
                    variance         = current_projection_correct_template.ReturnSumOfSquares( ) - powf(current_projection_correct_template.ReturnAverageOfRealValues( ), 2);

                    hipStreamWaitEvent(hipStreamPerThread, ref_correct_projection_is_free_Event, 0);

                    //// TO THE GPU ////
                    d_current_projection_correct_template.CopyHostToDevice( );
                    d_current_projection_correct_template.AddConstant(-average_of_reals);
                    d_current_projection_correct_template.MultiplyByConstant(rsqrtf(variance));
                    //current_projection_2.AddGaussianNoise(10.0f);

                    d_current_projection_correct_template.ClipInto(&d_padded_reference_correct, 0, false, 0, 0, 0, 0);
                    // Zeroing the central pixel is probably not doing anything useful...
                    // d_current_projection_other.ZeroCentralPixel( );
                    // FIXME is this necessary for each projection?
                    hipEventRecord(ref_correct_projection_is_free_Event, hipStreamPerThread);

                    d_padded_reference_correct.ForwardFFT(false);

                    d_padded_reference_correct.BackwardFFTAfterComplexConjMul(d_current_projection_image.complex_values_16f, true);
                    d_padded_reference_wrong.BackwardFFTAfterComplexConjMul(d_current_projection_image.complex_values_16f, true);

                    this->MipPixelWise(__float2half_rn(current_psi_tm), __float2half_rn(global_euler_search_tm.list_of_search_parameters[current_search_position_tm][1]), __float2half_rn(global_euler_search_tm.list_of_search_parameters[current_search_position_tm][0]), view_counter);

                    hipEventRecord(ref_wrong_projection_is_free_Event, hipStreamPerThread);
                    ccc_counter++;
                    total_number_of_cccs_calculated++;

                    if ( ccc_counter % 10 == 0 ) {
                        this->UpdateSums(my_stats_ac, d_sum1_ac, d_sumSq1_ac, view_counter);
                        this->UpdateSums(my_stats_cc, d_sum1_cc, d_sumSq1_cc, view_counter);
                    }

                    if ( ccc_counter % 100 == 0 ) {
                        d_sum2_ac.AddImage(d_sum1_ac);
                        d_sum1_ac.Zeros( );

                        d_sum2_cc.AddImage(d_sum1_cc);
                        d_sum1_cc.Zeros( );

                        d_sumSq2_ac.AddImage(d_sumSq1_ac);
                        d_sumSq1_ac.Zeros( );

                        d_sumSq2_cc.AddImage(d_sumSq1_cc);
                        d_sumSq1_cc.Zeros( );
                    }

                    if ( ccc_counter % 10000 == 0 ) { // TODO DEBUF AFTER
                        d_sum3_ac.AddImage(d_sum2_ac);
                        d_sum2_ac.Zeros( );

                        d_sum3_cc.AddImage(d_sum2_cc);
                        d_sum2_cc.Zeros( );

                        d_sumSq3_ac.AddImage(d_sumSq2_ac);
                        d_sumSq2_ac.Zeros( );

                        d_sumSq3_cc.AddImage(d_sumSq2_cc);
                        d_sumSq2_cc.Zeros( );
                    }

                    current_projection_other.is_in_real_space            = false;
                    current_projection_correct_template.is_in_real_space = false;
                    d_padded_reference_correct.is_in_real_space          = true;
                    d_padded_reference_wrong.is_in_real_space            = true;

                    hipEventRecord(current_tm_is_done, hipStreamPerThread);

                } // loop over tm psi angles
            } //loop over tm euler sphere position

            current_projection_image.is_in_real_space   = false;
            d_current_projection_image.is_in_real_space = false;

            hipEventRecord(image_projection_is_free_Event, hipStreamPerThread);
            hipStreamWaitEvent(hipStreamPerThread, current_tm_is_done, 0);

            wxPrintf("worker %i finished view %i total number %d\n\n", ReturnThreadNumberOfCurrentThread( ), view_counter, ccc_counter);

            // starting from here DEBUG PRIORITY

            this->UpdateSums(my_stats_ac, d_sum1_ac, d_sumSq1_ac, view_counter);
            this->UpdateSums(my_stats_cc, d_sum1_cc, d_sumSq1_cc, view_counter);
            // starting from here DEBUG PRIORITY

            d_sum2_ac.AddImage(d_sum1_ac); // AddImageBySlice not working; memory messed up; try to use single slice images instead;
            d_sumSq2_ac.AddImage(d_sumSq1_ac);

            d_sum3_ac.AddImage(d_sum2_ac);
            d_sumSq3_ac.AddImage(d_sumSq2_ac);

            d_sum2_cc.AddImage(d_sum1_cc);
            d_sumSq2_cc.AddImage(d_sumSq1_cc);

            d_sum3_cc.AddImage(d_sum2_cc);
            d_sumSq3_cc.AddImage(d_sumSq2_cc);
            // to here DEBUG

            /*
            d_sum2_ac.AddImage(d_sum1_ac);
            d_sumSq2_ac.AddImage(d_sumSq1_ac);

            d_sum3_ac.AddImage(d_sum2_ac);
            d_sumSq3_ac.AddImage(d_sumSq2_ac);

            d_sum2_cc.AddImage(d_sum1_cc);
            d_sumSq2_cc.AddImage(d_sumSq1_cc);

            d_sum3_cc.AddImage(d_sum2_cc);
            d_sumSq3_cc.AddImage(d_sumSq2_cc);
*/
            this->WriteMipToImage(view_counter);

            if ( ReturnThreadNumberOfCurrentThread( ) == 0 ) {
                current_correlation_position_sampled_view++;
                if ( current_correlation_position_sampled_view > total_correlation_positions_sampled_view_per_thread )
                    current_correlation_position_sampled_view = total_correlation_positions_sampled_view_per_thread;
                my_progress->Update(current_correlation_position_sampled_view); // move progress bar to inside loop more informative this way?
            }
            hipEventRecord(current_view_is_done, hipStreamPerThread); //testing here
            d_max_intensity_projection_ac.QuickAndDirtyWriteSlices(wxString::Format("check_gpu_run/d_mip_view_%i.mrc", view_counter).ToStdString( ), 1, d_sum1_ac.dims.z);
        } // loop over sampled views psi angles
        // there seems to be problem between the outer for loops connection
    } // loop over sampled views euler sphere position

    exit(0);
    hipStreamWaitEvent(hipStreamPerThread, current_view_is_done, 0);

    cudaErr(hipStreamSynchronize(hipStreamPerThread));

    cudaErr(hipFree(my_peaks_ac));
    cudaErr(hipFree(my_stats_ac));
    cudaErr(hipFree(my_new_peaks_ac));
    cudaErr(hipFree(my_peaks_cc));
    cudaErr(hipFree(my_stats_cc));
    cudaErr(hipFree(my_new_peaks_cc));
}

void TemplateSnrRatioCore::MipPixelWise(__half psi, __half theta, __half phi, const int view_counter) {

    precheck
            // N*
            d_padded_reference_correct.ReturnLaunchParamtersLimitSMs(5.f, 1024);
    d_padded_reference_wrong.ReturnLaunchParamtersLimitSMs(5.f, 1024);

    UpdateMipPixelWiseKernel<<<d_padded_reference_correct.gridDims, d_padded_reference_correct.threadsPerBlock, 0, hipStreamPerThread>>>((__half*)d_padded_reference_correct.real_values_16f, my_peaks_ac, (int)d_padded_reference_correct.real_memory_allocated, psi, theta, phi, my_stats_ac, my_new_peaks_ac, view_counter);
    UpdateMipPixelWiseKernel<<<d_padded_reference_wrong.gridDims, d_padded_reference_wrong.threadsPerBlock, 0, hipStreamPerThread>>>((__half*)d_padded_reference_wrong.real_values_16f, my_peaks_cc, (int)d_padded_reference_wrong.real_memory_allocated, psi, theta, phi, my_stats_cc, my_new_peaks_cc, view_counter);

    postcheck
}

__global__ void
UpdateMipPixelWiseKernel(__half* correlation_output, __half2* my_peaks, const int numel, __half psi, __half theta, __half phi, __half2* my_stats, __half2* my_new_peaks, const int view_counter) {

    //	Peaks tmp_peak;

    for ( int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x ) {

        const __half  half_val = correlation_output[i];
        const __half2 input    = __half2half2(half_val * __half(10000.0)); // cc*10000
        const __half2 mulVal   = __halves2half2((__half)1.0, half_val);

        my_stats[i + view_counter * numel] = __hfma2(input, mulVal, my_stats[i + view_counter * numel]); // TODO check top_K script

        if ( half_val > __low2half(my_peaks[i + view_counter * numel]) ) {
            //				tmp_peak.mip = half_val;
            my_peaks[i + view_counter * numel]     = __halves2half2(half_val, psi);
            my_new_peaks[i + view_counter * numel] = __halves2half2(theta, phi);
        }
    }
    //
}

__global__ void WriteMipToImageKernel(const __half2*, const __half2* my_new_peaks, const int, hipfftReal*, const int);

void TemplateSnrRatioCore::WriteMipToImage(int view_counter) {

    precheck
            dim3 threadsPerBlock = dim3(1024, 1, 1);
    dim3         gridDims        = dim3((d_max_intensity_projection_ac.real_memory_allocated / d_max_intensity_projection_ac.dims.z + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1); // CHECKME FIXME does gridDims matter? Do I really need to divide by z?

    WriteMipToImageKernel<<<gridDims, threadsPerBlock, 0, hipStreamPerThread>>>(my_peaks_ac, my_new_peaks_ac, d_max_intensity_projection_ac.real_memory_allocated / d_max_intensity_projection_ac.dims.z, d_max_intensity_projection_ac.real_values_gpu, view_counter);
    WriteMipToImageKernel<<<gridDims, threadsPerBlock, 0, hipStreamPerThread>>>(my_peaks_cc, my_new_peaks_cc, d_max_intensity_projection_cc.real_memory_allocated / d_max_intensity_projection_ac.dims.z, d_max_intensity_projection_cc.real_values_gpu, view_counter);

    postcheck
}

__global__ void WriteMipToImageKernel(const __half2* my_peaks, const __half2* my_new_peaks, const int numel, hipfftReal* mip, const int view_counter) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    if ( x < numel ) {
        mip[x + view_counter * numel] = (hipfftReal)__low2float(my_peaks[x + view_counter * numel]);
    }
}

__global__ void UpdateSumsKernel(__half2* temp_my_stats, const int numel, hipfftReal* sum, hipfftReal* sq_sum, const int view_counter);

void TemplateSnrRatioCore::UpdateSums(__half2* temp_my_stats, GpuImage& sum, GpuImage& sq_sum, const int view_counter) {

    precheck
            dim3 threadsPerBlock = dim3(1024, 1, 1);
    dim3         gridDims        = dim3((sum.real_memory_allocated / sum.dims.z + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    UpdateSumsKernel<<<gridDims, threadsPerBlock, 0, hipStreamPerThread>>>(temp_my_stats, sum.real_memory_allocated / sum.dims.z, sum.real_values_gpu, sq_sum.real_values_gpu, view_counter);
    postcheck
}

__global__ void UpdateSumsKernel(__half2* temp_my_stats, const int numel, hipfftReal* sum, hipfftReal* sq_sum, const int view_counter) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if ( x < numel ) {

        sum[x + numel * view_counter]    = __fmaf_rn(0.0001f, __low2float(temp_my_stats[x + numel * view_counter]), sum[x + numel * view_counter]);
        sq_sum[x + numel * view_counter] = __fmaf_rn(0.0001f, __high2float(temp_my_stats[x + numel * view_counter]), sq_sum[x + numel * view_counter]);

        temp_my_stats[x + numel * view_counter] = __halves2half2((__half)0., (__half)0.);
    }
}
